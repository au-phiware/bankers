#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#define choke(ERR, EXIT, STMT, ...)         \
{   ERR = STMT;                             \
    if (ERR != hipSuccess) {               \
        fprintf(stderr, "Failed to ");      \
        fprintf(stderr, __VA_ARGS__);       \
        fprintf(stderr, ": %s.\n",          \
                hipGetErrorString(ERR));   \
        if (EXIT) exit(EXIT);               \
}   }
#define debugf(...) if (debug) fprintf(stderr, __VA_ARGS__)

#ifndef SEP
#define SEP "\n"
#endif

// CUDA Capability v1.1 can only handle 32bit numbers
// with atomicAdd
// length <= sizeof(banker_t) && length <= COUNT_MAX
#ifndef length
#define length 64
#endif
#if length > 32
// don't exceed 2^sizeof(count_t) bit length
typedef unsigned long long int banker_t;
typedef unsigned char count_t;
#define FMT "%llU"
#else
// (don't exceed 2^sizeof(count_t) bit length)
typedef unsigned int banker_t;
typedef unsigned char count_t;
#define FMT "%u"
#endif

#define sharedMemorySize (0x4000 - 0x10)
// binomial coeffiecient function
// (indexes into the binom table aka Pascal's triangle)
#define choose(N, Y) ((Y) > (N) / 2 ? binom[rowOffset(N) + (N) - (Y)] : binom[rowOffset(N) + (Y)])
// Index of the Xth row of the binom table
// Note: each row is folded
#define rowOffset(X) (((((X) + 1) * ((X) + 1)) / 4) - 1)
// Maximum number of inputs per block with the available sharedMemorySize
#define maxBlockWidth (((sharedMemorySize - sizeof(banker_t) * rowOffset(length + 1)) / length) / sizeof(count_t))
// The block height (y dimension) must be the max width of the binom table
#define blockHeight (length / 2)

// Total number of threads per block
int threads = 0;

// Debug flag
int debug = 0;
// The current device
int device = 0;
// Device properties if needed
hipDeviceProp_t *deviceProp = NULL;

__global__ void inverse (banker_t* io)
{
    __shared__ count_t count[maxBlockWidth][length];
    __shared__ banker_t binom[rowOffset(length + 1)];
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = threadIdx.y;
    banker_t b = io[x], i = 1;

    for (int row = 1; row <= length; row++, i <<= 1) {
        int j = rowOffset(row) + y;
        __syncthreads();

        if (y == 0) {
            // Count cardinality at each bit position
            count[threadIdx.x][row - 1] = b & i ? 1 : 0;
            if (row > 1)
                count[threadIdx.x][row - 1] += count[threadIdx.x][row - 2];

            // Every binom row starts (and ends) with 1
            binom[j] = 1;
        } else if (threadIdx.x == 0 && y <= row / 2) {
            // Compute every cell of binom for row
            // cell is the sum of the two cells below it
            int k = j - (row + 1) / 2 - 1;
            binom[j]  = binom[k];
            // Center cell of every other row is double the cell in the
            // previous row (so we skip the increment)
            if (!(row % 2 == 0 && y == row / 2))
                k++;
            binom[j] += binom[k];
        }
    }

    // short-circuit for zero (zero maps to zero)
    if (b == 0) return;

    __syncthreads();
    count_t c = count[threadIdx.x][length - 1];
#if length > 32 && __CUDA_ARCH__ < 120
    if (y == 0) {
        banker_t a = 0;
        unsigned int n = length;

        for (i = 1; n-- && c > 0; i <<= 1)
            if (b & i)
                --c, a += choose(length, c);
            else
                a += choose(n, c - 1);

        io[x] = a;
    }
#else
    io[x] = 0;
    count_t c_y = count[threadIdx.x][y];

    if (b & (1 << y))
        atomicAdd(io + x, choose(length, c_y - 1));
    else if (c_y < c)
        atomicAdd(io + x, choose(length - y - 1, c - c_y - 1));
#endif
}

void usage(char *argv) {
    printf("Usage: %s [--threads=count] [--skip=n] [--limit=m]\n", argv);
    exit(1);
}

/*
 * Parse command line arguments
 */
int parse(int argc, char ** argv, banker_t ** inputPtr)
{
    banker_t x;
    unsigned int i, skip = 0, limit = 0;
    char *line = NULL;
    size_t n = 0;

    for (i = 1; i < argc; i++) {
        if (strncmp("--threads=", argv[i], 10) == 0) {
            threads = atoi(&argv[i][10]);
        } else if (strncmp("--skip=", argv[i], 7) == 0) {
            skip = atoi(&argv[i][7]);
        } else if (strncmp("--limit=", argv[i], 8) == 0) {
            limit = atoi(&argv[i][8]);
        } else if (strncmp("--device", argv[i], 8) == 0) {
            device = atoi(&argv[i][8]);
        } else if (strncmp("--debug", argv[i], 7) == 0) {
            debug = 1;
        } else {
            usage(*argv);
        }
    }
    for (i = 0; (limit == 0 || i < limit + skip) && getline(&line, &n, stdin) != -1; i++) {
        x = (banker_t) strtoull(line, NULL, 10);
        if (i >= skip) {
            if (((i - skip) % 0x100) == 0) {
                *inputPtr = (banker_t *)realloc(*inputPtr, (i - skip + 0x100) * sizeof(banker_t));
                if (*inputPtr == NULL) {
                    fprintf(stderr, "Failed to (re)allocate host array!\n");
                    exit(EXIT_FAILURE);
                }
            }
            (*inputPtr)[i - skip] = x;
        }
    }
    if (line) free(line);

    return i - skip;
}

void setDevice() {
    hipError_t err = hipSuccess;
    if (device == 0) {
        choke(err, EXIT_FAILURE,
                hipGetDevice(&device),
                "get current device");
        debugf("Got current device, %d.\n", device);
    } else {
        int d;
        choke(err, EXIT_FAILURE,
                hipGetDevice(&d),
                "get current device");
        debugf("Got current device, %d.\n", d);
        if (d != device) {
            choke(err, EXIT_FAILURE,
                    hipSetDevice(device),
                    "set current device to %d", device);
            debugf("Set current device to %d.\n", device);
        }
    }
}

void getDeviceProperties() {
    hipError_t err = hipSuccess;
    if (deviceProp == NULL) {
        deviceProp = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
        choke(err, EXIT_FAILURE,
                hipGetDeviceProperties(deviceProp, device),
                "get device (%d) properties", device);
    }
}

void setBestThreadSize() {
    getDeviceProperties();
    threads = deviceProp->maxThreadsDim[0];
    debugf("Setting threads to %d.\n", threads);
    if (deviceProp->maxThreadsPerBlock / blockHeight < threads / blockHeight) {
        threads = deviceProp->maxThreadsPerBlock;
        debugf("Setting threads to %d since maxThreadsPerBlock=%d.\n", threads, deviceProp->maxThreadsPerBlock);
    }
}

/*
 * Main program accepts one parameter: the number of the row
 * of Pascal's triangle to print.
 */
int main (int argc, char ** argv)
{
    hipError_t err = hipSuccess;
    int blocks, asize, size = argc - 1;
    banker_t *harray = NULL;
    banker_t *darray = NULL;

    // Parse argv and return the number of inputs specified by the user
    size = parse(argc, argv, &harray);
    if (harray == NULL) {
        fprintf(stderr, "Failed to allocate host array!\n");
        exit(EXIT_FAILURE);
    }
    setDevice();

    // Find a good number of threads if none explicitly specified by the user
    if (threads < 1) setBestThreadSize();
    // The height of the block must accomodate the width of the binom table
    if (threads < blockHeight) {
        threads = blockHeight;
        debugf("Setting threads to %d since it must be at least the width of the binom table (%d).\n", threads, blockHeight);
    }
    // threads must be divisible by the block height
    threads /= blockHeight;
    // The total number of threads per block must fit available shared memory
    if (threads > maxBlockWidth) {
        threads = maxBlockWidth;
        debugf("Setting threads to %d due to the available shared memory (%d bytes).\n", threads, sharedMemorySize);
    }
    // The number of blocks must cover the input size
    blocks = (size + threads - 1) / threads;
    // The aligned size (exact multiple of threads)
    asize = blocks * threads;

    choke(err, EXIT_FAILURE,
            hipMalloc((void **)&darray, asize * sizeof(banker_t)),
            "allocate device array of %zu bytes",
            asize * sizeof(banker_t));

    choke(err, EXIT_FAILURE,
            hipMemset(darray, 0, asize * sizeof(banker_t)),
            "initialise device array");

    choke(err, EXIT_FAILURE,
            hipMemcpy(darray, harray, size * sizeof(banker_t), hipMemcpyHostToDevice),
            "copy array from host to device");

    debugf("Launching %d block%s of %d by %d threads...\n",
                blocks, blocks == 1 ? "" : "s", threads, blockHeight);
    dim3 t (threads, blockHeight);
    inverse<<<blocks, t>>>(darray);
    choke(err, EXIT_FAILURE,
            hipGetLastError(),
            "launch inverse kernel");

    choke(err, EXIT_FAILURE,
            hipDeviceSynchronize(),
            "complete inverse kernel");

    choke(err, EXIT_FAILURE,
            hipMemcpy(harray, darray, size * sizeof(banker_t), hipMemcpyDeviceToHost),
            "copy array from device to host");

    choke(err, EXIT_FAILURE,
            hipFree(darray),
            "free device array");

    printf(FMT, *harray);
    for(int i = 1; i < size; i++) {
        printf(SEP FMT, harray[i]);
    }
    printf("\n");

    free(harray);
    free(deviceProp);

    choke(err, EXIT_FAILURE,
            hipDeviceReset(),
            "deinitialize the device");

    return 0;
}
