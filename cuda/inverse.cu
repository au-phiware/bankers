#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#define choke(ERR, EXIT, STMT, ...)         \
{   ERR = STMT;                             \
    if (ERR != hipSuccess) {               \
        fprintf(stderr, "Failed to ");      \
        fprintf(stderr, __VA_ARGS__);       \
        fprintf(stderr, ": %s.\n",          \
                hipGetErrorString(ERR));   \
        if (EXIT) exit(EXIT);               \
}   }

#ifndef SEP
#define SEP " "
#endif

// CUDA Capability v1.1 can only handle 32bit numbers
// with atomicAdd
// length <= sizeof(banker_t) && length <= COUNT_MAX
#ifndef length
#define length 64
#endif
#if length > 32
// don't exceed 2^sizeof(count_t) bit length
typedef unsigned long long int banker_t;
typedef unsigned char count_t;
#define FMT "%llU"
#else
// (don't exceed 2^sizeof(count_t) bit length)
typedef unsigned int banker_t;
typedef unsigned char count_t;
#define FMT "%u"
#endif

#define sharedMemorySize (0x4000 - 0x10)
// binomial coeffiecient function
// (indexes into the binom table aka Pascal's triangle)
#define choose(N, Y) ((Y) > (N) / 2 ? binom[rowOffset(N) + (N) - (Y)] : binom[rowOffset(N) + (Y)])
// Index of the Xth row of the binom table
// Note: each row is folded
#define rowOffset(X) (((((X) + 1) * ((X) + 1)) / 4) - 1)
// Maximum number of inputs per block with the available sharedMemorySize
#define maxBlockWidth (((sharedMemorySize - sizeof(banker_t) * rowOffset(length + 1)) / length) / sizeof(count_t))

int threads = 256;

__global__ void inverse (banker_t* io)
{
    __shared__ count_t count[maxBlockWidth][length];
    __shared__ banker_t binom[rowOffset(length + 1)];
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = threadIdx.y;
    banker_t b = io[x], i = 1;

    for (int row = 1; row <= length; row++, i <<= 1) {
        int j = rowOffset(row) + y;
        __syncthreads();

        if (y == 0) {
            // Count cardinality at each bit position
            count[threadIdx.x][row - 1] = b & i ? 1 : 0;
            if (row > 1)
                count[threadIdx.x][row - 1] += count[threadIdx.x][row - 2];

            // Every binom row starts (and ends) with 1
            binom[j] = 1;
        } else if (threadIdx.x == 0 && y <= row / 2) {
            // Compute every cell of binom for row
            // cell is the sum of the two cells below it
            int k = j - (row + 1) / 2 - 1;
            binom[j]  = binom[k];
            // Center cell of every other row is double the cell in the
            // previous row (so we skip the increment)
            if (!(row % 2 == 0 && y == row / 2))
                k++;
            binom[j] += binom[k];
        }
    }

    // short-circuit for zero (zero maps to zero)
    if (b == 0) return;

    __syncthreads();
    count_t c = count[threadIdx.x][length - 1];
#if length > 32 && __CUDA_ARCH__ < 120
    if (y == 0) {
        banker_t a = 0;
        unsigned int n = length;

        for (i = 1; n-- && c > 0; i <<= 1)
            if (b & i)
                --c, a += choose(length, c);
            else
                a += choose(n, c - 1);

        io[x] = a;
    }
#else
    io[x] = 0;
    count_t c_y = count[threadIdx.x][y];

    if (b & (1 << y))
        atomicAdd(io + x, choose(length, c_y - 1));
    else if (c_y < c)
        atomicAdd(io + x, choose(length - y - 1, c - c_y - 1));
#endif
}

/*
 * Parse command line arguments
 */
int parse(int argc, char ** argv, banker_t * input)
{
    banker_t x;
    int skip = 1;

    if (argc < 2)
    {
        printf("Usage: %s [-threads=count] n ...\n", argv[0]);
        exit(1);
    }

    for (int i = 1; i < argc; i++) {
        if (strncmp("-threads=", argv[i], 9) == 0) {
            skip++;
            threads = atoi(&argv[i][9]);
        } else {
            x = (banker_t) strtoull(argv[i], NULL, 10);
            input[i - skip] = x;
        }
    }

    return argc - skip;
}

void setBestThreadSize() {}

/*
 * Main program accepts one parameter: the number of the row
 * of Pascal's triangle to print.
 */
int main (int argc, char ** argv)
{
    hipError_t err = hipSuccess;
    int blocks, asize, size = argc - 1;
    banker_t *harray = (banker_t *)malloc(size * sizeof(banker_t));
    banker_t *darray = NULL;

    if (harray == NULL) {
        fprintf(stderr, "Failed to allocate host array!\n");
        exit(EXIT_FAILURE);
    }

    size = parse(argc, argv, harray);
    if (threads < 1) setBestThreadSize();
    if (threads < length / 2) threads = length / 2;
    threads /= length;
    if (threads > maxBlockWidth) threads = maxBlockWidth;
    blocks = (size + threads - 1) / threads;
    asize = blocks * size;

    choke(err, EXIT_FAILURE,
            hipMalloc((void **)&darray, asize * sizeof(banker_t)),
            "allocate device array of %zu bytes",
            asize * sizeof(banker_t));

    choke(err, EXIT_FAILURE,
            hipMemset(darray, 0, asize * sizeof(banker_t)),
            "initialise device array");

    choke(err, EXIT_FAILURE,
            hipMemcpy(darray, harray, size * sizeof(banker_t), hipMemcpyHostToDevice),
            "copy array from host to device");

    dim3 t (threads, length);
    inverse<<<blocks, t>>>(darray);
    choke(err, EXIT_FAILURE,
            hipGetLastError(),
            "launch inverse kernel");

    choke(err, EXIT_FAILURE,
            hipDeviceSynchronize(),
            "complete inverse kernel");

    choke(err, EXIT_FAILURE,
            hipMemcpy(harray, darray, size * sizeof(banker_t), hipMemcpyDeviceToHost),
            "copy array from device to host");

    choke(err, EXIT_FAILURE,
            hipFree(darray),
            "free device array");

    printf(FMT, *harray);
    for(int i = 1; i < size; i++) {
        printf(SEP FMT, harray[i]);
    }
    printf("\n");

    free(harray);

    choke(err, EXIT_FAILURE,
            hipDeviceReset(),
            "deinitialize the device");

    return 0;
}
