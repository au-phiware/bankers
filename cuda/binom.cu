// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include <helper_functions.h>

#define rowOffset(X) ((((X) - 1) * ((X) - 1)) / 4)

__global__ void binom(unsigned long *table, const int n)
{
    __shared__ unsigned long cache[0x100];
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    for (int row = 2; row <= n; row++) {
        int i = rowOffset(row) + col;
        __syncthreads();
        if (col == 0) {
            table[i] = cache[i & 0xFF] = row;
        } else if (col < row / 2) {
            int j = rowOffset(row - 1) + col - 1;
            cache[i & 0xFF]  = cache[j & 0xFF];
            if (!(row % 2 == 0 && col == row / 2 - 1))
                j++;
            cache[i & 0xFF] += cache[j & 0xFF];
            table[i] = cache[i & 0xFF];
        }
    }
}

/*
 * Maximum number of rows (specified in program args).
 */
static unsigned int length;

/*
 * Parse command line arguments
 */
unsigned int parse(int argc, char ** argv)
{
    unsigned int i;
    if (argc != 2)
    {
        printf("Usage: %s n\n", argv[0]);
        exit(1);
    }
    i = atoi(argv[1]);
    if (i > 67) {
        fprintf(stderr, "Warning: %U is too big, results will be affected by integer overflow.");
    }
    return i;
}

/*
 * Main program accepts one parameter: the number of the row
 * of Pascal's triangle to print.
 */
int main (int argc, char ** argv)
{
    length = parse(argc, argv);

    hipError_t err = hipSuccess;
    size_t size = rowOffset(length + 1) * sizeof(unsigned long);
    unsigned long *table = (unsigned long *)malloc(size);
    unsigned long *d_table = NULL;
    err = hipMalloc((void **)&d_table, size);

    if (table == NULL) {
        fprintf(stderr, "Failed to allocate host table!\n");
        exit(EXIT_FAILURE);
    }
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device table: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    binom<<<1, length + 1>>>(d_table, length);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch binom kernel: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(table, d_table, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy table from device to host: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_table);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device table: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    unsigned int i = rowOffset(length);
    printf("1 %lU", *table);
    for (++i; i < rowOffset(length + 1); i++) {
        printf("%s%lU", table[i]);
    }
    if (length % 2 == 0) i--;
    for (--i; i >= rowOffset(length); i--) {
        printf(" %lU", table[i]);
    }
    printf(" 1\n");

    free(table);
    return 0;
}
