// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#define choke(ERR, EXIT, MSG, ...)          \
{   ERR = __VA_ARGS__;                      \
    if (ERR != hipSuccess) {               \
        fprintf(stderr,                     \
                "Failed to " MSG ": %s.\n", \
                hipGetErrorString(ERR));   \
        if (EXIT) exit(EXIT);               \
}   }
#define ulsize(S) ((S) * sizeof(unsigned long))

#define SEP "\n"
#define length 64
#define setBit(B) (B |= (1ul << (length - 1)))
#define rowOffset(X) (((X) * ((X) + 1)) / 2 - 1)
#define choose(N, C) binom[rowOffset(N) + (C)]

int threads = 256;
__constant__ unsigned long binom[rowOffset(length + 1)];

__global__ void compute (unsigned long* a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (a[i] == 0) return;

    unsigned long b = 0;
    unsigned int c = 1, n = length, j = rowOffset(length);
    unsigned long e = a[i] - 1;

    while (binom[j + 1] <= e) {
        ++j, ++c;
        e -= binom[j];
    }

    do {
        j -= n;
        if (e == 0 || binom[j] > e)
            --j, --c, setBit(b);
        else
            e -= binom[j];
    } while (--n && c && ((b >>= 1) || 1));
    b >>= n;

    a[i] = b;
}

__global__ void binom_kernel(unsigned long *table, const int n)
{
    __shared__ unsigned long cache[0x100];
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    for (int row = 1; row <= n; row++) {
        int i = rowOffset(row) + col;
        __syncthreads();
        if (col == 0 || col == row) {
            table[i] = cache[i & 0xff] = 1;
        } else if (col < row) {
            int j = i - row;
            table[i] = cache[i & 0xff] = cache[j & 0xff] + cache[(j - 1) & 0xff];
        }
    }
}
void initBinomTable()
{
    hipError_t err = hipSuccess;
    size_t size = ulsize(rowOffset(length + 1));
    unsigned long *hbinom = (unsigned long *)malloc(size);
    unsigned long *dbinom = NULL;

    if (hbinom == NULL) {
        fprintf(stderr, "Failed to allocate host binom table!\n");
        exit(EXIT_FAILURE);
    }

    choke(err, EXIT_FAILURE, "allocate device binom table",
            hipMalloc((void **)&dbinom, size));

    binom_kernel<<<1, length + 1>>>(dbinom, length);
    choke(err, EXIT_FAILURE, "launch binom kernel",
            hipGetLastError());

    choke(err, EXIT_FAILURE, "copy binom table from device to host",
            hipMemcpy(hbinom, dbinom, size, hipMemcpyDeviceToHost));

    choke(err, EXIT_FAILURE, "free device binom table",
            hipFree(dbinom));

    choke(err, EXIT_FAILURE, "copy binom table from host to device (constant memory)",
            hipMemcpyToSymbol(HIP_SYMBOL(binom), hbinom, size));

    free(hbinom);
}

/*
 * Parse command line arguments
 */
int parse(int argc, char ** argv, unsigned long * input)
{
    unsigned long x;
    int skip = 1;

    if (argc < 2)
    {
        printf("Usage: %s [-threads=count] n ...\n", argv[0]);
        exit(1);
    }

    for (int i = 1; i < argc; i++) {
        if (strncmp("-threads=", argv[i], 9) == 0) {
            skip++;
            threads = atoi(&argv[i][9]);
        } else {
            x = strtoul(argv[i], NULL, 10);
            input[i - skip] = x;
        }
    }

    return argc - skip;
}

/*
 * Main program accepts one parameter: the number of the row
 * of Pascal's triangle to print.
 */
int main (int argc, char ** argv)
{
    hipError_t err = hipSuccess;
    int blocks, asize, size = argc - 1;
    unsigned long *harray = (unsigned long *)malloc(ulsize(size));
    unsigned long *darray = NULL;

    initBinomTable();

    if (harray == NULL) {
        fprintf(stderr, "Failed to allocate host array!\n");
        exit(EXIT_FAILURE);
    }

    size = parse(argc, argv, harray);
    blocks = (size + threads - 1) / threads;
    asize = blocks * size;

    choke(err, EXIT_FAILURE, "allocate device array",
            hipMalloc((void **)&darray, ulsize(asize)));

    choke(err, EXIT_FAILURE, "initialise device array",
            hipMemset(darray, 0, ulsize(asize)));

    choke(err, EXIT_FAILURE, "copy array from host to device",
            hipMemcpy(darray, harray, ulsize(size), hipMemcpyHostToDevice));

    compute<<<blocks, threads>>>(darray);
    choke(err, EXIT_FAILURE, "launch compute kernel",
            hipGetLastError());

    choke(err, EXIT_FAILURE, "copy array from device to host",
            hipMemcpy(harray, darray, ulsize(size), hipMemcpyDeviceToHost));

    choke(err, EXIT_FAILURE, "free device array",
            hipFree(darray));

    printf("%lU", *harray);
    for(int i = 1; i < size; i++) {
        printf(SEP "%lU", harray[i]);
    }
    printf("\n");
    
    free(harray);

    choke(err, EXIT_FAILURE, "deinitialize the device",
            hipDeviceReset());

    return 0;
}
