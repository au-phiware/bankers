// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#define choke(ERR, EXIT, STMT, ...)         \
{   ERR = STMT;                             \
    if (ERR != hipSuccess) {               \
        fprintf(stderr, "Failed to ");      \
        fprintf(stderr, __VA_ARGS__);       \
        fprintf(stderr, ": %s.\n",          \
                hipGetErrorString(ERR));   \
        if (EXIT) exit(EXIT);               \
}   }
#define debugf(...) if (debug) fprintf(stderr, __VA_ARGS__)

// Debug flag
int debug = 0;

#ifndef SEP
#define SEP "\n"
#endif

// CUDA Capability v1.1 can only handle 32bit numbers
// with atomicAdd
// length <= sizeof(banker_t) && length <= COUNT_MAX
#ifndef length
#define length 64
#endif
#if length > 32
// don't exceed 2^sizeof(count_t) bit length
typedef unsigned long long int banker_t;
typedef unsigned char count_t;
#define FMT "%llU"
#define setBit(B) (B |= (1ull << (length - 1)))
#else
// (don't exceed 2^sizeof(count_t) bit length)
typedef unsigned int banker_t;
typedef unsigned char count_t;
#define FMT "%u"
#define setBit(B) (B |= (1 << (length - 1)))
#endif

// binomial coeffiecient function
// (indexes into the binom table aka Pascal's triangle)
#define choose(N, C) binom[rowOffset(N) + (C)]
// Index of the Xth row of the binom table
#define rowOffset(X) (((X) * ((X) + 1)) / 2 - 1)

int threads = 256;
__constant__ banker_t binom[rowOffset(length + 1)];

__global__ void compute (banker_t* a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (a[i] == 0) return;

    banker_t b = 0;
    unsigned int c = 1, n = length, j = rowOffset(length);
    banker_t e = a[i] - 1;

    while (binom[j + 1] <= e) {
        ++j, ++c;
        e -= binom[j];
    }

    do {
        j -= n;
        if (e == 0 || binom[j] > e)
            --j, --c, setBit(b);
        else
            e -= binom[j];
    } while (--n && c && ((b >>= 1) || 1));
    b >>= n;

    a[i] = b;
}

__global__ void binom_kernel(banker_t *table, const int n)
{
    __shared__ banker_t cache[0x100];
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    for (int row = 1; row <= n; row++) {
        int i = rowOffset(row) + col;
        __syncthreads();
        if (col == 0 || col == row) {
            table[i] = cache[i & 0xff] = 1;
        } else if (col < row) {
            int j = i - row;
            table[i] = cache[i & 0xff] = cache[j & 0xff] + cache[(j - 1) & 0xff];
        }
    }
}
void initBinomTable()
{
    hipError_t err = hipSuccess;
    size_t size = (rowOffset(length + 1)) * sizeof(banker_t);
    banker_t *hbinom = (banker_t *)malloc(size);
    banker_t *dbinom = NULL;

    if (hbinom == NULL) {
        fprintf(stderr, "Failed to allocate host binom table!\n");
        exit(EXIT_FAILURE);
    }

    choke(err, EXIT_FAILURE,
            hipMalloc((void **)&dbinom, size),
            "allocate device binom table");

    debugf("Launching a block of %d threads...\n", length + 1);
    binom_kernel<<<1, length + 1>>>(dbinom, length);
    choke(err, EXIT_FAILURE,
            hipGetLastError(),
            "launch binom kernel");

    choke(err, EXIT_FAILURE,
            hipMemcpy(hbinom, dbinom, size, hipMemcpyDeviceToHost),
            "copy binom table from device to host");

    choke(err, EXIT_FAILURE,
            hipFree(dbinom),
            "free device binom table");

    choke(err, EXIT_FAILURE,
            hipMemcpyToSymbol(HIP_SYMBOL(binom), hbinom, size),
            "copy binom table from host to device (constant memory)");

    free(hbinom);
}

void usage(char *argv) {
    printf("Usage: %s [--threads=count] [--skip=n] [--limit=m]\n", argv);
    exit(1);
}

/*
 * Parse command line arguments
 */
int parse(int argc, char ** argv, banker_t ** inputPtr)
{
    banker_t x;
    unsigned int i, skip = 0, limit = 0;
    char *line = NULL;
    size_t n = 0;

    for (i = 1; i < argc; i++) {
        if (strncmp("--threads=", argv[i], 10) == 0) {
            threads = atoi(&argv[i][10]);
        } else if (strncmp("--skip=", argv[i], 7) == 0) {
            skip = atoi(&argv[i][7]);
        } else if (strncmp("--limit=", argv[i], 8) == 0) {
            limit = atoi(&argv[i][8]);
        } else if (strncmp("--debug", argv[i], 7) == 0) {
            debug = 1;
        } else {
            usage(*argv);
        }
    }
    for (i = 0; (limit == 0 || i < limit + skip) && getline(&line, &n, stdin) != -1; i++) {
        x = (banker_t) strtoull(line, NULL, 10);
        if (i >= skip) {
            if (((i - skip) % 0x100) == 0) {
                *inputPtr = (banker_t *)realloc(*inputPtr, (i - skip + 0x100) * sizeof(banker_t));
                if (*inputPtr == NULL) {
                    fprintf(stderr, "Failed to (re)allocate host array!\n");
                    exit(EXIT_FAILURE);
                }
            }
            (*inputPtr)[i - skip] = x;
        }
    }
    if (line) free(line);

    return i - skip;
}

/*
 * Main program accepts one parameter: the number of the row
 * of Pascal's triangle to print.
 */
int main (int argc, char ** argv)
{
    hipError_t err = hipSuccess;
    int blocks, asize, size = argc - 1;
    banker_t *harray = NULL;
    banker_t *darray = NULL;

    initBinomTable();

    size = parse(argc, argv, &harray);
    if (harray == NULL) {
        fprintf(stderr, "Failed to allocate host array!\n");
        exit(EXIT_FAILURE);
    }

    blocks = (size + threads - 1) / threads;
    asize = blocks * threads;

    choke(err, EXIT_FAILURE,
            hipMalloc((void **)&darray, asize * sizeof(banker_t)),
            "allocate device array");

    choke(err, EXIT_FAILURE,
            hipMemset(darray, 0, asize * sizeof(banker_t)),
            "initialise device array");

    choke(err, EXIT_FAILURE,
            hipMemcpy(darray, harray, size * sizeof(banker_t), hipMemcpyHostToDevice),
            "copy array from host to device");

    debugf("Launching %d block%s of %d thread%s...\n",
                blocks, blocks == 1 ? "" : "s", threads, threads == 1 ? "" : "s");
    compute<<<blocks, threads>>>(darray);
    choke(err, EXIT_FAILURE,
            hipGetLastError(),
            "launch compute kernel");

    choke(err, EXIT_FAILURE,
            hipMemcpy(harray, darray, size * sizeof(banker_t), hipMemcpyDeviceToHost),
            "copy array from device to host");

    choke(err, EXIT_FAILURE,
            hipFree(darray),
            "free device array");

    printf(FMT, *harray);
    for(int i = 1; i < size; i++) {
        printf(SEP FMT, harray[i]);
    }
    printf("\n");

    free(harray);

    choke(err, EXIT_FAILURE,
            hipDeviceReset(),
            "deinitialize the device");

    return 0;
}
