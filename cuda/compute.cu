// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#define choke(ERR, EXIT, STMT, ...)         \
{   ERR = STMT;                             \
    if (ERR != hipSuccess) {               \
        fprintf(stderr, "compute: ");       \
        fprintf(stderr, "Failed to ");      \
        fprintf(stderr, __VA_ARGS__);       \
        fprintf(stderr, ": %s.\n",          \
                hipGetErrorString(ERR));   \
        if (EXIT) exit(EXIT);               \
}   }
#define debugf(...) if (debug) fprintf(stderr, "compute: " __VA_ARGS__)

// Debug flag
int debug = 0;
// The current device
int device = 0;
// Device properties if needed
hipDeviceProp_t *deviceProp = NULL;

#ifndef SEP
#define SEP "\n"
#endif

// CUDA Capability v1.1 can only handle 32bit numbers
// with atomicAdd
// length <= sizeof(banker_t) && length <= COUNT_MAX
#ifndef length
#define length 64
#endif
#if length > 32
// don't exceed 2^sizeof(count_t) bit length
typedef unsigned long long int banker_t;
typedef unsigned char count_t;
#define FMT "%llU"
#define setBit(B) (B |= (1ull << (length - 1)))
#else
// (don't exceed 2^sizeof(count_t) bit length)
typedef unsigned int banker_t;
typedef unsigned char count_t;
#define FMT "%u"
#define setBit(B) (B |= (1 << (length - 1)))
#endif

// binomial coeffiecient function
// (indexes into the binom table aka Pascal's triangle)
#define choose(N, C) binom[rowOffset(N) + (C)]
// Index of the Xth row of the binom table
#define rowOffset(X) (((X) * ((X) + 1)) / 2 - 1)

// Total number of threads per block
int threads = 0;
// Maximum size that can be allocated or processed on device
int maxInputArraySize = 0;

__constant__ banker_t binom[rowOffset(length + 1)];

__global__ void compute (banker_t* a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (a[i] == 0) return;

    banker_t b = 0;
    unsigned int c = 1, n = length, j = rowOffset(length);
    banker_t e = a[i] - 1;

    while (binom[j + 1] <= e) {
        ++j, ++c;
        e -= binom[j];
    }

    do {
        j -= n;
        if (e == 0 || binom[j] > e)
            --j, --c, setBit(b);
        else
            e -= binom[j];
    } while (--n && c && ((b >>= 1) || 1));
    b >>= n;

    a[i] = b;
}

__global__ void binom_kernel(banker_t *table, const int n)
{
    __shared__ banker_t cache[0x100];
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    for (int row = 1; row <= n; row++) {
        int i = rowOffset(row) + col;
        __syncthreads();
        if (col == 0 || col == row) {
            table[i] = cache[i & 0xff] = 1;
        } else if (col < row) {
            int j = i - row;
            table[i] = cache[i & 0xff] = cache[j & 0xff] + cache[(j - 1) & 0xff];
        }
    }
}
void initBinomTable()
{
    hipError_t err = hipSuccess;
    size_t size = (rowOffset(length + 1)) * sizeof(banker_t);
    banker_t *hbinom = (banker_t *)malloc(size);
    banker_t *dbinom = NULL;

    if (hbinom == NULL) {
        fprintf(stderr, "Failed to allocate host binom table!\n");
        exit(EXIT_FAILURE);
    }

    choke(err, 0,
            hipMalloc((void **)&dbinom, size),
            "allocate device binom table");
    if (err != hipSuccess) {
        size_t memFree, memTot;
        choke(err, EXIT_FAILURE,
                hipMemGetInfo(&memFree, &memTot),
                "get device memory information");
        debugf("Available device memory is %zu bytes.\n", memFree);
        exit(EXIT_FAILURE);
    }

    debugf("Launching a block of %d threads...\n", length + 1);
    binom_kernel<<<1, length + 1>>>(dbinom, length);
    choke(err, EXIT_FAILURE,
            hipGetLastError(),
            "launch binom kernel");

    choke(err, EXIT_FAILURE,
            hipMemcpy(hbinom, dbinom, size, hipMemcpyDeviceToHost),
            "copy binom table from device to host");

    choke(err, EXIT_FAILURE,
            hipFree(dbinom),
            "free device binom table");

    choke(err, EXIT_FAILURE,
            hipMemcpyToSymbol(HIP_SYMBOL(binom), hbinom, size),
            "copy binom table from host to device (constant memory)");

    free(hbinom);
}

void setDevice() {
    hipError_t err = hipSuccess;
    if (device == 0) {
        choke(err, EXIT_FAILURE,
                hipGetDevice(&device),
                "get current device");
    } else {
        int d;
        choke(err, EXIT_FAILURE,
                hipGetDevice(&d),
                "get current device");
        if (d != device)
            choke(err, EXIT_FAILURE,
                    hipSetDevice(device),
                    "set current device to %d", device);
    }
}

void getDeviceProperties() {
    hipError_t err = hipSuccess;
    if (deviceProp == NULL) {
        deviceProp = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
        choke(err, EXIT_FAILURE,
                hipGetDeviceProperties(deviceProp, device),
                "get device (%d) properties", device);
    }
}

void setBestThreadSize() {
    getDeviceProperties();
    threads = deviceProp->maxThreadsDim[0];
}

void setMaxInputArraySize() {
    if (maxInputArraySize == 0) {
        hipError_t err = hipSuccess;
        size_t memFree, memTot;
        setDevice();

        choke(err, EXIT_FAILURE,
                hipMemGetInfo(&memFree, &memTot),
                "get device memory information");
        maxInputArraySize = memFree / sizeof(banker_t) / 2; // Don't be greedy, take half
        debugf("Setting maxInputArraySize to %d with available device memory at %zu bytes.\n", maxInputArraySize, memFree);
    }
}

void usage(char *argv) {
    printf("Usage: %s [--threads=count] [--skip=n] [--limit=m]\n", argv);
    exit(1);
}

/*
 * Parse command line arguments
 */
int parse(int argc, char ** argv, banker_t ** inputPtr, int offset)
{
    banker_t x;
    unsigned int i, skip = 0, limit = 0;
    char *line = NULL;
    size_t n = 0;

    for (i = 1; i < argc; i++) {
        if (strncmp("--threads=", argv[i], 10) == 0) {
            threads = atoi(&argv[i][10]);
        } else if (strncmp("--skip=", argv[i], 7) == 0 && offset == 0) {
            skip = atoi(&argv[i][7]);
        } else if (strncmp("--limit=", argv[i], 8) == 0) {
            limit = atoi(&argv[i][8]);
        } else if (strncmp("--device", argv[i], 8) == 0) {
            device = atoi(&argv[i][8]);
        } else if (strncmp("--debug", argv[i], 7) == 0) {
            debug = 1;
        } else {
            usage(*argv);
        }
    }
    setMaxInputArraySize();
    for (i = 0; (limit == 0 || i < limit - offset + skip) && i < maxInputArraySize && getline(&line, &n, stdin) != -1; i++) {
        x = (banker_t) strtoull(line, NULL, 10);
        if (i >= skip) {
            if (((i - skip) % 0x100) == 0) {
                *inputPtr = (banker_t *)realloc(*inputPtr, (i - skip + 0x100) * sizeof(banker_t));
                if (*inputPtr == NULL) {
                    fprintf(stderr, "Failed to (re)allocate host array!\n");
                    exit(EXIT_FAILURE);
                }
            }
            (*inputPtr)[i - skip] = x;
        }
    }
    if (line) free(line);

    return i - skip;
}

/*
 * Main program accepts one parameter: the number of the row
 * of Pascal's triangle to print.
 */
int main (int argc, char ** argv)
{
    hipError_t err = hipSuccess;
    int blocks, asize, size, done = 0;
    banker_t *harray = NULL;
    banker_t *darray = NULL;

    initBinomTable();

    size = parse(argc, argv, &harray, done);
    setDevice();
    if (threads < 1)
        setBestThreadSize();

    do if (size > 0)
    {
        if (harray == NULL) {
            fprintf(stderr, "Failed to allocate host array!\n");
            exit(EXIT_FAILURE);
        }

        blocks = (size + threads - 1) / threads;
        asize = blocks * threads;

        if (darray == NULL)
            choke(err, EXIT_FAILURE,
                    hipMalloc((void **)&darray, asize * sizeof(banker_t)),
                    "allocate device array");

        choke(err, EXIT_FAILURE,
                hipMemset(darray, 0, asize * sizeof(banker_t)),
                "initialise device array");

        choke(err, EXIT_FAILURE,
                hipMemcpy(darray, harray, size * sizeof(banker_t), hipMemcpyHostToDevice),
                "copy array from host to device");

        debugf("Launching %d block%s of %d thread%s...\n",
                    blocks, blocks == 1 ? "" : "s", threads, threads == 1 ? "" : "s");
        compute<<<blocks, threads>>>(darray);
        choke(err, EXIT_FAILURE,
                hipGetLastError(),
                "launch compute kernel");

        choke(err, EXIT_FAILURE,
                hipMemcpy(harray, darray, size * sizeof(banker_t), hipMemcpyDeviceToHost),
                "copy array from device to host");

        printf(FMT, *harray);
        for(int i = 1; i < size; i++) {
            printf(SEP FMT, harray[i]);
        }
        printf("\n");

        done += size;
    } while ((size = parse(argc, argv, &harray, done)) > 0);

    if (darray != NULL)
        choke(err, EXIT_FAILURE,
                hipFree(darray),
                "free device array");
    if (harray != NULL)
        free(harray);
    if (deviceProp != NULL)
        free(deviceProp);

    choke(err, EXIT_FAILURE,
            hipDeviceReset(),
            "deinitialize the device");

    return 0;
}
